#include "hip/hip_runtime.h"
/*This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/    
#include "str_sort.h"

using namespace std;

int main(int ac, char **av)
{
    string file_name;
	thrust::device_vector<char> delimiter(1);
	delimiter[0] = ' ';
	string usage = "Usage : nvsort [-t FIELD_SEPARATOR] [-n NUMERIC_SORT] [-k KEYS] FILE";
	vector<unsigned int> keys;
	vector<unsigned int> keys_numeric;
	uint64_t chunk_sz = 800000000; // segment size, 800MB is ok for GTX 1080
	bool reverse = 0;
	map<size_t, vector<string> > files_to_merge;
	thrust::host_vector<char, thrust::cuda::experimental::pinned_allocator<char> > file_buffer(chunk_sz);
	thrust::device_vector<char> device_file_buffer(chunk_sz);
	thrust::device_vector<char> device_file_buffer_out(chunk_sz);	
	
	std::clock_t start1 = std::clock();	
	
	if (ac <= 1) {
        cout << usage << endl;
        exit(1);
    };


    for(auto i = 1; i < ac; i++) {
        if(strcmp(av[i],"-t") == 0) {
            if(i+1 < ac) {
                delimiter[0] = av[i+1][0];
				i++;
            }
            else {
                cout << usage << endl;
                exit(1);
            };
        }
        else if(strcmp(av[i],"-k") == 0) {
			if(i+1 < ac) {
				if(av[i+1][strlen(av[i+1])-1] == 'n') {
					keys_numeric.push_back(1);
					av[i+1][strlen(av[i+1])-1] = '\0';
				}
				else 
					keys_numeric.push_back(0);
				keys.push_back(atoi(av[i+1]));
				i++;
			}	
			else {
                cout << usage << endl;
                exit(1);
            };
		}
        else if(strcmp(av[i],"-r") == 0) {
            reverse = 1;
        }	
		else 	
			file_name = av[i];			
    };
	if(file_name.length() == 0)
	{
      cout << usage << endl;
      exit(1);
    };		

    if(keys.size() == 0) { //sort by entire strings
    	keys.push_back(1);
    	keys_numeric.push_back(0);
    	delimiter[0] = '\n';
    };	
	
	//cout << "Sep " << delimiter[0] << " file " << file_name << endl;
	//for(int i = 0; i < keys.size(); i++)
	//	cout << "Key " << keys[i] << " " << keys_numeric[i] << endl;
	
		
	auto file_size = filesize(file_name.c_str());	
	ifstream f(file_name.c_str(), ios::binary);	

	//Sort phase	
	unsigned int tot_read = 0;
	bool first = 1;

	for(unsigned int k = 0; k <= file_size/chunk_sz + 1 && f; k++) {	
		
		f.read(file_buffer.data(), chunk_sz);
		uint64_t read_cnt = f.gcount();
		if(read_cnt == chunk_sz) {
			int j = 0;
			while(file_buffer[chunk_sz-j-1] != '\n')
				j++;
			read_cnt = read_cnt-j;				
			f.seekg(-j, f.cur);
			//cout << "sort rewind " << j << endl;
		};	
		tot_read = tot_read + read_cnt;
		
		thrust::copy(file_buffer.begin(), file_buffer.begin() + read_cnt, device_file_buffer.begin());		
		str_sort(keys, keys_numeric, device_file_buffer, device_file_buffer_out, delimiter, read_cnt, 0, first, reverse);		
		thrust::copy(device_file_buffer_out.begin(), device_file_buffer_out.begin() + read_cnt, file_buffer.begin());		

		auto file_out = file_name + ".sorted";	
		if(file_size/chunk_sz > 0) {
			file_out = file_out + '.' + to_string(k);
			files_to_merge[read_cnt].push_back(file_out);			
		};		
		fstream sorted_file(file_out.c_str(),ios::out|ios::binary);
		sorted_file.write((char *)file_buffer.data(), read_cnt);		
		sorted_file.close();			

	};
	f.close();
	//std::cout<< "sort phase time: " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<  '\n';    	

	//merge phase
	unsigned int k = 0;	
	while(files_to_merge.size() > 1) {
		string first_file, second_file;
		auto it = files_to_merge.begin();
		first_file = it->second[0];
		if(it->second.size() > 1) {		
			second_file = it->second[1];
		}
		else {
			it++;
			second_file = it->second[0];
		};
		
		auto file_out = file_name + ".merged." + to_string(k++);	
		//cout << "files " << first_file << " " << second_file << " " << file_out << endl;
		ifstream f1(first_file.c_str(), ios::binary);	
		ifstream f2(second_file.c_str(), ios::binary);	 
		fstream sorted_file(file_out.c_str(),ios::out|ios::binary);       
        size_t total_written = 0;
        size_t cnt1, cnt2, rewind;	
        int j;
        uint64_t read_cnt = 0;
        bool file_to_rewind;          

		while (f1 && f2) {		   
		   
           f1.read(file_buffer.data(), chunk_sz/2);
           read_cnt = f1.gcount();
		   if (read_cnt == 0)
		   	  break;           
           if(read_cnt == chunk_sz/2) {
              j = 0;
              while(file_buffer[read_cnt-j-1] != '\n')
			     j++;
		      cnt1 = read_cnt-j;				
		      f1.seekg(-j, f1.cur); 
		   }
		   else
		   	   cnt1 = read_cnt;
           thrust::copy(file_buffer.begin(), file_buffer.begin() + cnt1, device_file_buffer.begin());		 

	   	   f2.read(file_buffer.data(), chunk_sz/2);
           read_cnt = f2.gcount();
		   if (read_cnt == 0)
		   	  break;                      
           if(read_cnt == chunk_sz/2) {
              j = 0;
              while(file_buffer[read_cnt-j-1] != '\n')
			     j++;
		      cnt2 = read_cnt-j;				
	          f2.seekg(-j, f2.cur);               
		   }
	       else
		      cnt2 = read_cnt;
           thrust::copy(file_buffer.begin(), file_buffer.begin() + cnt2, device_file_buffer.begin() + cnt1);	
           auto sorted_sz = str_merge(keys, keys_numeric, device_file_buffer, device_file_buffer_out, delimiter,
                                    cnt1 + cnt2, cnt1, file_to_rewind, reverse);	
           //cout << "sorted sz " << sorted_sz << endl; 
           if((!f1 && file_to_rewind == 1) || (!f2 && file_to_rewind == 0))   {                
              sorted_sz =  cnt1 + cnt2;    	
           };   
       	   thrust::copy(device_file_buffer_out.begin(), device_file_buffer_out.begin() + sorted_sz, file_buffer.begin());		
       	   sorted_file.write((char *)file_buffer.data(), sorted_sz);		
       	   total_written = total_written + sorted_sz;
       	   rewind = cnt1 + cnt2 - sorted_sz;
       	   //cout << "rewind " << file_to_rewind << " " << rewind << endl;
       	   if(file_to_rewind == 0) {
       	   	  if(rewind > 0) {
       	   		 if(!f1)
       	   	        f1.clear();       	   	
  	             f1.seekg(-rewind, f1.cur);  	             
       	   	  }
       	   }
       	   else {
       	   	  if(rewind > 0) {
       	   	     if(!f2) 
       	   	        f2.clear();
                 f2.seekg(-rewind, f2.cur);       	   
       	   	  }
       	   }
		}; 

		if(f1) {
		   do {	
			   f1.read(file_buffer.data(), chunk_sz/2);
	           read_cnt = f1.gcount();
	           sorted_file.write((char *)file_buffer.data(), read_cnt);		
	           //cout << "wrote " << read_cnt  << " from f1 " << endl;
	          }
	       while(f1);       
		}
		else if(f2) {
		   do {	
			   f2.read(file_buffer.data(), chunk_sz/2);
	           read_cnt = f2.gcount();
	           sorted_file.write((char *)file_buffer.data(), read_cnt);		
	           //cout << "wrote " << read_cnt  << " from f2 " << endl;
              } 
           while(f2);
		};
		total_written = total_written + read_cnt;


		f1.close();
		f2.close();
		sorted_file.close();		
		it = files_to_merge.begin();
		it->second.erase(it->second.begin());	
		if(it->second.size() > 0) {		
			it->second.erase(it->second.begin());
			if(it->second.size() == 0)
               files_to_merge.erase(it);
		}
		else {
			files_to_merge.erase(it++);
			it->second.erase(it->second.begin());
			if(it->second.size() == 0)
               files_to_merge.erase(it);
		};

		files_to_merge[total_written].push_back(file_out);			
		//remove(first_file.c_str());
		//remove(second_file.c_str());
	};
	
	auto file_merged = file_name + ".merged." + to_string(k-1);	
	auto file_final = file_name + ".sorted";	
	rename(file_merged.c_str(), file_final.c_str());

	//std::cout<< "merge phase time: " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<  '\n';    	
    return 0;

}
