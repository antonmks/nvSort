#include "hip/hip_runtime.h"
/*This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/    
#include "str_sort.h"

using namespace std;
using namespace mgpu;


void str_sort(std::vector<unsigned int>& keys, std::vector<unsigned int>& keys_numeric, thrust::device_vector<char>& device_file_buffer,
		      thrust::device_vector<char>& device_file_buffer_out, thrust::device_vector<char>& delimiter, size_t read_cnt,
		      size_t first_offset, bool& file_to_rewind, bool reverse)
{

    standard_context_t context(0);	    
	thrust::counting_iterator<unsigned int> begin(0);
	char h_delimiter = delimiter[0];
	std::clock_t start1 = std::clock();			
 
	auto begin_keys = thrust::make_zip_iterator(thrust::make_tuple(device_file_buffer.begin(), thrust::counting_iterator<int>(1)));
    auto end_keys = thrust::make_zip_iterator(thrust::make_tuple(device_file_buffer.begin() + read_cnt, thrust::counting_iterator<int>(read_cnt)));
    auto cnt = thrust::count(device_file_buffer.begin(), device_file_buffer.begin() + read_cnt,'\n');
    thrust::device_vector<int> nl_pos(cnt);
    auto res_keys = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_discard_iterator(), nl_pos.begin()));
    thrust::copy_if(begin_keys, end_keys, res_keys, count_newlines());  
	
	//for(int z = 0; z < nl_pos.size(); z++)
	//	cout << "nlpos " << nl_pos[z] << endl;
	thrust::device_vector<unsigned int> key_num(1);	
	if(nl_pos.back() == read_cnt) {
	    nl_pos.erase(nl_pos.end()-1);	    
	};
	nl_pos.insert(nl_pos.begin(), 0);

	//for(int z = 0; z < nl_pos.size(); z++)
	//	cout << "nlpos fin  " << nl_pos[z] << endl;


	thrust::device_vector<unsigned int> perm(cnt);	
	thrust::sequence(perm.begin(), perm.end(), 0, 1);			
	
	for(unsigned int i = 0; i < keys.size(); i++) {

		key_num[0] = keys[keys.size()-1-i];
		if(keys_numeric[keys.size()-1-i] == 1) {
			thrust::device_vector<double> device_key_double(cnt);	
			thrust::device_vector<double> device_key_tmp(cnt);	
					
			gpu_atof at((const char*)thrust::raw_pointer_cast(device_file_buffer.data()), thrust::raw_pointer_cast(device_key_double.data()),               
						(const unsigned int *)thrust::raw_pointer_cast(key_num.data()), (const char*)thrust::raw_pointer_cast(delimiter.data()),
						(const int*)thrust::raw_pointer_cast(nl_pos.data()));
			thrust::for_each(begin, begin + cnt, at);						
			thrust::gather(perm.begin(), perm.end(), device_key_double.begin(), device_key_tmp.begin());
		    if(reverse) {	
			    thrust::stable_sort_by_key(device_key_tmp.begin(), device_key_tmp.end(), perm.begin(), thrust::greater<double>());
			}
			else {
				thrust::stable_sort_by_key(device_key_tmp.begin(), device_key_tmp.end(), perm.begin());	
			}    
		}
		else {
			
			thrust::device_vector<char*> field_pos(cnt);	
			thrust::device_vector<char*> field_pos_tmp(cnt);			
			gpu_find_pos fp((const char*)thrust::raw_pointer_cast(device_file_buffer.data()), (char**)thrust::raw_pointer_cast(field_pos.data()),               
						(const unsigned int *)thrust::raw_pointer_cast(key_num.data()), (const char*)thrust::raw_pointer_cast(delimiter.data()),
						(const int*)thrust::raw_pointer_cast(nl_pos.data()));
			thrust::for_each(begin, begin + cnt, fp);			
			thrust::gather(perm.begin(), perm.end(), field_pos.begin(), field_pos_tmp.begin());
			if(reverse) {
				sort_str_desc f(h_delimiter);	
				mergesort(thrust::raw_pointer_cast(field_pos_tmp.data()), thrust::raw_pointer_cast(perm.data()), perm.size(), f, context); 
			}	
			else {
				sort_str f(h_delimiter);	
				mergesort(thrust::raw_pointer_cast(field_pos_tmp.data()), thrust::raw_pointer_cast(perm.data()), perm.size(), f, context); 
			}
			//hipDeviceSynchronize();
		};	
	}	

	thrust::device_vector<int> nl_len(cnt);
	thrust::transform(nl_pos.begin()+1, nl_pos.end(), nl_pos.begin(), nl_len.begin(),  thrust::minus<int>());
	nl_len[cnt-1] = read_cnt - nl_pos[cnt-1];
	//cout << "last len " << nl_len[cnt-1] << " " << read_cnt << " " << nl_pos[cnt-1] << endl;
		
	thrust::device_vector<int> nl_len_g(cnt);
    thrust::device_vector<int> nl_len_pos(cnt);
	thrust::gather(perm.begin(), perm.end(), nl_len.begin(), nl_len_g.begin());	
	thrust::exclusive_scan(nl_len_g.begin(), nl_len_g.end(), nl_len_g.begin());	
    thrust::scatter(nl_len_g.begin(), nl_len_g.end(), perm.begin(), nl_len_pos.begin());	
    interval_scatter(thrust::raw_pointer_cast(device_file_buffer.data()), read_cnt, thrust::raw_pointer_cast(nl_pos.data()), nl_pos.size(), thrust::raw_pointer_cast(nl_len_pos.data()), thrust::raw_pointer_cast(device_file_buffer_out.data()), context);        
}	


size_t str_merge(std::vector<unsigned int>& keys, std::vector<unsigned int>& keys_numeric, thrust::device_vector<char>& device_file_buffer,
		         thrust::device_vector<char>& device_file_buffer_out, thrust::device_vector<char>& delimiter,
		         size_t read_cnt, size_t first_offset, bool& file_to_rewind, bool reverse)
{
	
	thrust::counting_iterator<unsigned int> begin(0);
	char h_delimiter = delimiter[0];
	std::clock_t start1 = std::clock();		
    
	auto begin_keys = thrust::make_zip_iterator(thrust::make_tuple(device_file_buffer.begin(), thrust::counting_iterator<int>(1)));
    auto end_keys = thrust::make_zip_iterator(thrust::make_tuple(device_file_buffer.begin() + read_cnt, thrust::counting_iterator<int>(read_cnt)));
    auto cnt = thrust::count(device_file_buffer.begin(), device_file_buffer.begin() + read_cnt,'\n');
    auto first_cnt = thrust::count(device_file_buffer.begin(), device_file_buffer.begin() + first_offset,'\n');
    //cout << "test lines " << cnt << " " << first_cnt << " " << read_cnt << endl;
    
    thrust::device_vector<int> nl_pos(cnt);
    auto res_keys = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_discard_iterator(), nl_pos.begin()));
    thrust::copy_if(begin_keys, end_keys, res_keys, count_newlines());  
	
	thrust::device_vector<unsigned int> key_num(1);	
	if(nl_pos.back() == read_cnt) {
	    nl_pos.erase(nl_pos.end()-1);	    
	};
	nl_pos.insert(nl_pos.begin(), 0);


	thrust::device_vector<unsigned int> perm(cnt);	
    thrust::device_vector<unsigned int> perm_tmp(cnt);  
	thrust::sequence(perm.begin(), perm.end(), 0, 1);		
    thrust::device_vector<char*> nl_pos_char(cnt);    

    set_address ff((const char*)thrust::raw_pointer_cast(device_file_buffer.data()),
                   (const unsigned int*)thrust::raw_pointer_cast(nl_pos.data()), 
                   thrust::raw_pointer_cast(nl_pos_char.data()));
    thrust::for_each(begin, begin + cnt, ff);    

    thrust::device_vector<char*> field_pos_tmp(cnt);      
    thrust::device_vector<unsigned int> d_keys(keys.size());
    thrust::copy(keys.data(), keys.data()+keys.size(), d_keys.begin());
    thrust::device_vector<unsigned int> d_keys_numeric(keys.size());    
    thrust::copy(keys_numeric.data(), keys_numeric.data()+keys.size(), d_keys_numeric.begin());

    if(reverse) {
	    compare_fields_desc f(h_delimiter, thrust::raw_pointer_cast(d_keys.data()), thrust::raw_pointer_cast(d_keys_numeric.data()), keys.size());
	    //merge(nl_pos_char.data(), perm.begin(), first_cnt,
	    //      nl_pos_char.data() + first_cnt,  perm.begin() + first_cnt, cnt-first_cnt,
	    //      field_pos_tmp.data(), perm_tmp.begin(), f, context);	
	}
	else{
	    compare_fields f(h_delimiter, thrust::raw_pointer_cast(d_keys.data()), thrust::raw_pointer_cast(d_keys_numeric.data()), keys.size());
	    //merge(nl_pos_char.data(), perm.begin(), first_cnt,
	    //      nl_pos_char.data() + first_cnt,  perm.begin() + first_cnt, cnt-first_cnt,
	    //      field_pos_tmp.data(), perm_tmp.begin(), f, context);		
	}
	
	thrust::device_vector<int> nl_len(cnt);
	thrust::transform(nl_pos.begin()+1, nl_pos.end(), nl_pos.begin(), nl_len.begin(),  thrust::minus<int>());
	nl_len[cnt-1] = read_cnt - nl_pos[cnt-1];

	thrust::device_vector<int> nl_len_g(cnt);
    thrust::device_vector<int> nl_len_pos(cnt);
	thrust::gather(perm_tmp.begin(), perm_tmp.end(), nl_len.begin(), nl_len_g.begin());	
	thrust::exclusive_scan(nl_len_g.begin(), nl_len_g.end(), nl_len_g.begin());	
    thrust::scatter(nl_len_g.begin(), nl_len_g.end(), perm_tmp.begin(), nl_len_pos.begin());	

	//for(int z = 0; z < 10; z++)
	//	cout << "nl pos  " << nl_pos[z] << " " << nl_len_pos[z] << endl;
    //interval_scatter(thrust::raw_pointer_cast(device_file_buffer.data()), read_cnt, thrust::raw_pointer_cast(nl_pos.data()), nl_pos.size(), 
    //	                                      thrust::raw_pointer_cast(nl_len_pos.data()), thrust::raw_pointer_cast(device_file_buffer_out.data()), context);        
    auto new_pos1 = nl_len_pos[first_cnt-1];
    auto new_pos2 = nl_len_pos[cnt-1];    
    //cout << "pos1 pos 2 " << new_pos1 << " " << new_pos2 << endl;
    if(new_pos1 < new_pos2) 
    {
        file_to_rewind = 1;
        return new_pos1 + nl_len[first_cnt-1];	
    }
    else
    {
   	    file_to_rewind = 0;
        return new_pos2 + nl_len.back();	        
    }    
}	


uint64_t filesize(const char* filename)
{
    ifstream in(filename, ios::binary | ios::ate);
	if(!in) {
		cout << "Could not open file " << filename << endl;
		exit(0);
	};	
    return in.tellg();
}

